#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// #include <lcutil.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "datatypes.h"

#define ERR(XX,YY) d_error[(YY)*(d_snd->n)+(XX)]


__global__ void recvKernel(sendtype *d_snd, recvtype *d_rec, double *d_u, double *d_u_old){
    #define SRC(XX,YY) d_u_old[(YY)*(d_snd->n+2)+(XX)]
    #define DST(XX,YY) d_u[(YY)*(d_snd->n+2)+(XX)]

    // double xLeft = -1.0;
    // double yBottom = -1.0;
    // double xLeft_local, yBottom_local;
    // int n_local, m_local, coords[2];

    // double deltaX = (2.0)/(d_snd->n-1);
    // double deltaY = (2.0)/(d_snd->m-1);

    // double cx = 1.0/(deltaX*deltaX);
    // double cy = 1.0/(deltaY*deltaY);
    // double cc = -2.0*cx-2.0*cy-(d_snd->alpha);

    SRC(0,0) += 1.0;
    // double *temp;
    // temp = d_u;
    // d_u = d_u_old;
    // d_u_old = temp;
    // d_rec->elem1 = DST(0,0);
    // d_rec->elem2 = DST(1,0);    
}

__global__ void coordCalc(sendtype *d_snd, double *fZ_squared){
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if(index < d_snd->n){
        fZ_squared[index] = -1.0 + index*d_snd->delta;
        fZ_squared[index] = fZ_squared[index]*fZ_squared[index];
    }
}

__global__ void jacobi(sendtype *d_snd, double *fXsquared, double *fYsquared, double *d_u_old, double *d_u, double *d_error){
    #define SRC(XX,YY) d_u_old[(YY)*(d_snd->n+2)+(XX)]
    #define DST(XX,YY) d_u[(YY)*(d_snd->n+2)+(XX)]

    int xIndex = threadIdx.x + blockDim.x*blockIdx.x + 1;
    int yIndex = threadIdx.y + blockDim.y*blockIdx.y + 1;
    double f, updateVal;

    if(xIndex < (d_snd->n + 1) && yIndex < (d_snd->m + 1)){
        f = -d_snd->alpha*(1.0-fXsquared[xIndex-1])*(1.0-fYsquared[yIndex-1]) - 2.0*(2.0-fXsquared[xIndex-1]-fYsquared[yIndex-1]);
        updateVal = (	(SRC(xIndex-1,yIndex) + SRC(xIndex+1,yIndex))*d_snd->cx +
                        (SRC(xIndex,yIndex-1) + SRC(xIndex,yIndex+1))*d_snd->cy +
                        SRC(xIndex,yIndex)*d_snd->cc - f
                    )/d_snd->cc;
        DST(xIndex,yIndex) = SRC(xIndex,yIndex) - d_snd->relax*updateVal;
        ERR(xIndex-1,yIndex-1) = updateVal*updateVal;
    }
}

int main(){
    sendtype *snd;
    snd = (sendtype *) malloc(sizeof(sendtype));

    scanf("%d,%d", &(snd->n), &(snd->m));
    scanf("%lf", &(snd->alpha));
    scanf("%lf", &(snd->relax));
    scanf("%lf", &(snd->tol));
    scanf("%d", &(snd->mits));
    printf("-> %d, %d, %g, %g, %g, %d\n", snd->n, snd->m, snd->alpha, snd->relax, snd->tol, snd->mits);
    snd->delta = 2.0/(snd->n-1);
    snd->cx = 1.0/(snd->delta*snd->delta);
    snd->cy = 1.0/(snd->delta*snd->delta);
    snd->cc = -2.0*snd->cx-2.0*snd->cy-snd->alpha;

    sendtype *d_snd;
    hipMalloc((void **) &d_snd, sizeof(sendtype));
    hipMemcpy(d_snd, snd, sizeof(sendtype), hipMemcpyHostToDevice);

    recvtype *rec, *d_rec;
    rec = (recvtype *) malloc(sizeof(recvtype));
    hipMalloc((void **) &d_rec, sizeof(recvtype));

    double *d_u, *d_u_old, *d_fXsquared, *d_fYsquared, *d_error;    
    hipError_t err = hipMalloc((void **) &d_u, (snd->n + 2)*(snd->m + 2)*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_u_old, (snd->n + 2)*(snd->m + 2)*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_fXsquared, snd->n*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_fYsquared, snd->m*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_error, snd->n*snd->m*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert for error array: %s\n", hipGetErrorString(err));
    }
    hipMemset(d_u, 0, (snd->n + 2)*(snd->m + 2)*sizeof(double));
    hipMemset(d_u_old, 0, (snd->n + 2)*(snd->m + 2)*sizeof(double));
    hipMemset(d_fXsquared, 0, snd->n*sizeof(double));
    hipMemset(d_fYsquared, 0, snd->m*sizeof(double));
    hipMemset(d_error, 0, snd->m*snd->n*sizeof(double));

    // I for sure will have 128 threads per block
    // So we now wish to find how many blocks are necessary for
    // dividing our problem size's *side* by 128
    int threadNum = 128;
    int blocksNum = ceil((double)snd->n/(double)threadNum);
    printf("threadNum = %d, blocksNum = %d\n", threadNum, blocksNum);
    coordCalc<<<blocksNum, threadNum>>>(d_snd, d_fXsquared);
    coordCalc<<<blocksNum, threadNum>>>(d_snd, d_fYsquared);
    // For the actual arrays, I choose 256 threads per block
    // in a 16x16 cartesian fashion. So now I need to find how
    // many blocks I need per side to have a 2D block grid
    dim3 threadsPerBlock(16, 16);
    blocksNum = ceil((double)snd->n/16.0);
    dim3 blocksInGrid(blocksNum, blocksNum);
    double *error = (double *)malloc(snd->n*snd->m*sizeof(double));
    double *temp;

    int iterationCount = 0;
    double error_all = 15.0;
    
    while(iterationCount < snd->mits && error_all > snd->tol){
        jacobi<<<blocksInGrid, threadsPerBlock>>>(d_snd, d_fXsquared, d_fYsquared, d_u_old, d_u, d_error);
        hipMemcpy(error, d_error, snd->n*snd->m*sizeof(double), hipMemcpyDeviceToHost);
        for(int i = 0; i < snd->n*snd->m; i++){
            error_all += error[i]; 
        }
        error_all = sqrt(error_all)/(snd->n*snd->m);
        iterationCount++;
        temp = d_u;
        d_u = d_u_old;
        d_u_old = temp;
    }

    printf("Iterations: %d\nResidual: %g\n", iterationCount, error_all);

    hipFree(d_u);
    hipFree(d_u_old);
    hipFree(d_fXsquared);
    hipFree(d_fYsquared);
    hipFree(d_snd);
    hipFree(d_rec);
    hipFree(d_error);
    free(snd);
    free(rec);
    free(error);
    return 0;
}
