#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
// #include <lcutil.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <math.h>
#include "datatypes.h"
#include <omp.h>

#define ERR(XX,YY) d_error[(YY)*(d_snd->n)+(XX)]


__global__ void coordCalc(sendtype *d_snd, double *fZ_squared){
    int index = threadIdx.x + blockDim.x*blockIdx.x;
    if(index < d_snd->n){
        fZ_squared[index] = -1.0 + index*d_snd->delta;
        fZ_squared[index] = fZ_squared[index]*fZ_squared[index];
    }
}

__global__ void jacobi(sendtype *d_snd, double *fXsquared, double *fYsquared, double *d_u_old, double *d_u, double *d_error){
    #define SRC(XX,YY) d_u_old[(YY)*(d_snd->n+2)+(XX)]
    #define DST(XX,YY) d_u[(YY)*(d_snd->n+2)+(XX)]

    int xIndex = threadIdx.x + blockDim.x*blockIdx.x + 1;
    int yIndex = threadIdx.y + blockDim.y*blockIdx.y + 1;
    double f, updateVal;

    if(xIndex < (d_snd->n + 1) && yIndex < (d_snd->m + 1)){
        f = -d_snd->alpha*(1.0-fXsquared[xIndex-1])*(1.0-fYsquared[yIndex-1]) - 2.0*(2.0-fXsquared[xIndex-1]-fYsquared[yIndex-1]);
        updateVal = (	(SRC(xIndex-1,yIndex) + SRC(xIndex+1,yIndex))*d_snd->cx +
                        (SRC(xIndex,yIndex-1) + SRC(xIndex,yIndex+1))*d_snd->cy +
                        SRC(xIndex,yIndex)*d_snd->cc - f
                    )/d_snd->cc;
        DST(xIndex,yIndex) = SRC(xIndex,yIndex) - d_snd->relax*updateVal;
        ERR(xIndex-1,yIndex-1) = updateVal*updateVal;
    }
}

__global__ void reduceError(double *d_error){
  int step_size = 1;
  int number_of_threads = 1024; // We'll ALWAYS START with this many active PER BLOCK

  int index;
  int fst, snd;

	while (number_of_threads > 0)
	{
    if (threadIdx.x < number_of_threads) // still alive?
    {
      index = threadIdx.x + number_of_threads*blockIdx.x;
      fst = index * step_size * 2;
      snd = fst + step_size;
      d_error[fst] += d_error[snd];
    }

    step_size *= 2; 
		number_of_threads = number_of_threads/2;
    if(threadIdx.x == 0 && number_of_threads == 0){ 
      // DONE, COPYING TO FIRST NUMBLOCKS 
      // OF D_ERROR ARRAY
      d_error[blockIdx.x] = d_error[2*blockIdx.x*blockDim.x];
    }
    __syncthreads();
	}
}

int main(){
    sendtype *snd;
    snd = (sendtype *) malloc(sizeof(sendtype));

    scanf("%d,%d", &(snd->n), &(snd->m));
    scanf("%lf", &(snd->alpha));
    scanf("%lf", &(snd->relax));
    scanf("%lf", &(snd->tol));
    scanf("%d", &(snd->mits));
    printf("-> %d, %d, %g, %g, %g, %d\n", snd->n, snd->m, snd->alpha, snd->relax, snd->tol, snd->mits);
    snd->delta = 2.0/(snd->n-1);
    snd->cx = 1.0/(snd->delta*snd->delta);
    snd->cy = 1.0/(snd->delta*snd->delta);
    snd->cc = -2.0*snd->cx-2.0*snd->cy-snd->alpha;

    sendtype *d_snd, *d_snd_two;
    hipMalloc((void **) &d_snd, sizeof(sendtype));
    snd->m /= 2;
    hipMemcpy(d_snd, snd, sizeof(sendtype), hipMemcpyHostToDevice);
    snd->m *= 2;

    recvtype *rec, *d_rec;
    rec = (recvtype *) malloc(sizeof(recvtype));
    hipMalloc((void **) &d_rec, sizeof(recvtype));

    double *d_u, *d_u_old, *d_fXsquared, *d_fYsquared, *d_error; 
    double *d_u_two, *d_u_old_two, *d_fYsquared_two, *d_fXsquared_two, *d_error_two;
    double *fXsquared_temp, *fYsquared_temp;        
    fYsquared_temp = (double *)malloc(snd->m*sizeof(double));
    fXsquared_temp = (double *)malloc(snd->n*sizeof(double));
    hipError_t err = hipMalloc((void **) &d_u, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_u_old, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_fXsquared, snd->n*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_fYsquared, snd->m*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    int zeroPaddedMemory = pow(2, ceil(log2((snd->m/2)*snd->n))); 
    err = hipMalloc((void **) &d_error, zeroPaddedMemory*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert for error array: %s\n", hipGetErrorString(err));
    }
    hipMemset(d_u, 0, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    hipMemset(d_u_old, 0, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    hipMemset(d_fXsquared, 0, snd->n*sizeof(double));
    hipMemset(d_error, 0, zeroPaddedMemory*sizeof(double));
    hipMemset(d_fYsquared, 0, snd->m*sizeof(double));

    hipSetDevice(1);
    snd->m /= 2;
    hipMalloc((void **) &d_snd_two, sizeof(sendtype));
    hipMemcpy(d_snd_two, snd, sizeof(sendtype), hipMemcpyHostToDevice);
    snd->m *= 2;
    err = hipMalloc((void **) &d_fXsquared_two, snd->n*sizeof(double));
    if (err != hipSuccess){
		  fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_fYsquared_two, (snd->m/2)*sizeof(double));
    if (err != hipSuccess){
		  fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_u_two, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_u_old_two, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert: %s\n", hipGetErrorString(err));
    }
    err = hipMalloc((void **) &d_error_two, zeroPaddedMemory*sizeof(double));
    if (err != hipSuccess){
		fprintf(stderr, "GPUassert for error array: %s\n", hipGetErrorString(err));
    }
    hipMemset(d_u_two, 0, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    hipMemset(d_u_old_two, 0, (snd->n + 2)*(snd->m/2 + 2)*sizeof(double));
    hipMemset(d_fXsquared_two, 0, snd->n*sizeof(double));
    hipMemset(d_error_two, 0, zeroPaddedMemory*sizeof(double));
    hipMemset(d_fYsquared_two, 0, (snd->m/2)*sizeof(double));
    hipSetDevice(0);

    int threadNum = 128;
    int blocksNum = ceil((double)snd->n/(double)threadNum);
    

    // I for sure will have 128 threads per block
    // So we now wish to find how many blocks are necessary for
    // dividing our problem size's *side* by 128
    clock_t start = clock(), diff;    
    coordCalc<<<blocksNum, threadNum>>>(d_snd, d_fYsquared);
    coordCalc<<<blocksNum, threadNum>>>(d_snd, d_fXsquared);
    hipDeviceSynchronize();
    hipMemcpy(fYsquared_temp, d_fYsquared, snd->m*sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(fXsquared_temp, d_fXsquared, snd->n*sizeof(double), hipMemcpyDeviceToHost);
    hipSetDevice(1);    
    hipMemcpy(d_fYsquared_two, &fYsquared_temp[snd->m/2], (snd->m/2)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_fXsquared_two, fXsquared_temp, snd->n*sizeof(double), hipMemcpyHostToDevice);
    hipSetDevice(0);    
    free(fYsquared_temp);
    free(fXsquared_temp);

    // For the actual arrays, I choose 256 threads per block
    // in a 16x16 cartesian fashion. So now I need to find how
    // many blocks I need per side to have a 2D block grid
    dim3 threadsPerBlock(16, 16);
    blocksNum = ceil((double)snd->n/16.0);
    dim3 blocksInGrid(blocksNum, blocksNum);

    // For error reduction, we can treat the error array as one-dimensional
    // We can use the code that the professor sent pretty much as is, despite
    // not having one block. We'll just find the thread's global id and use that
    // to sum, and the error will be in the very first element of the array
    double *temp;
        
    int iterationCount = 0;
    double error_all = HUGE_VAL;
    double error_all_two;
    double *halo = (double *)malloc(snd->n*sizeof(double));

    threadNum = 1024;
    // I wish for each block to have 2048 elements of the array to reduce 
    blocksNum = zeroPaddedMemory/2048;
    while(iterationCount < snd->mits && error_all > snd->tol){
    	error_all = 0.0;
      jacobi<<<blocksInGrid, threadsPerBlock>>>(d_snd, d_fXsquared, d_fYsquared, d_u_old, d_u, d_error);
      hipSetDevice(1);
      jacobi<<<blocksInGrid, threadsPerBlock>>>(d_snd_two, d_fXsquared_two, d_fYsquared_two, d_u_old_two, d_u_two, d_error_two);
      hipDeviceSynchronize();
      hipSetDevice(0);
      hipDeviceSynchronize();
      reduceError<<<blocksNum,threadNum>>>(d_error);
      hipSetDevice(1);
      reduceError<<<blocksNum,threadNum>>>(d_error_two);
      hipDeviceSynchronize();
      hipSetDevice(0);
      hipDeviceSynchronize();
      do{
        if(blocksNum < 2048){
          break;
        }else{
          blocksNum /= 2048;
          reduceError<<<blocksNum, threadNum>>>(d_error);
          hipSetDevice(1);
          reduceError<<<blocksNum, threadNum>>>(d_error_two);
          hipDeviceSynchronize();
          hipSetDevice(0);
          hipDeviceSynchronize();
        }
      }while(blocksNum != 1);
      hipMemset(&d_error[blocksNum], 0, (2048 - blocksNum)*sizeof(double));
      reduceError<<<1,threadNum>>>(d_error);
      hipSetDevice(1);
      hipMemset(&d_error_two[blocksNum], 0, (2048 - blocksNum)*sizeof(double));
      reduceError<<<1,threadNum>>>(d_error_two);
      hipDeviceSynchronize();
      hipSetDevice(0); hipDeviceSynchronize();
      
      hipMemcpy(&error_all, &d_error[0], sizeof(double), hipMemcpyDeviceToHost);
      hipSetDevice(1);
      hipMemcpy(&error_all_two, &d_error_two[0], sizeof(double), hipMemcpyDeviceToHost);
      hipSetDevice(0);
      error_all += error_all_two;
      error_all = sqrt(error_all)/(snd->n*snd->m);
      
      temp = d_u;
      d_u = d_u_old;
      d_u_old = temp;
      
      temp = d_u_two;
      d_u_two = d_u_old_two;
      d_u_old_two = temp;

      iterationCount++;
      hipMemcpy(halo, &d_u_old[(snd->m/2)*(snd->n + 2) + 1],snd->n*sizeof(double), hipMemcpyDeviceToHost);
      hipSetDevice(1);
      hipMemcpy(&d_u_old_two[1], halo, snd->n*sizeof(double), hipMemcpyHostToDevice);
      hipMemcpy(halo, &d_u_old_two[(snd->n + 2) + 1], snd->n*sizeof(double), hipMemcpyDeviceToHost);
      hipSetDevice(0);
      hipMemcpy(halo, &d_u_old[(snd->m/2 + 1)*(snd->n + 2) + 1], snd->n*sizeof(double), hipMemcpyHostToDevice);

      blocksNum = zeroPaddedMemory/2048;
    }
    diff = clock() - start;
    int msec = diff * 1000 / CLOCKS_PER_SEC;
    printf("Iterations: %d\nResidual: %g\n", iterationCount, error_all);
    printf("Time taken %d seconds %d milliseconds\n", msec/1000, msec%1000);

    hipFree(d_u);
    hipFree(d_u_old);
    hipFree(d_fXsquared);
    hipFree(d_fYsquared);
    hipFree(d_snd);
    hipFree(d_rec);
    hipFree(d_error);

    hipSetDevice(1);
    hipFree(d_u_two);
    hipFree(d_u_old_two);
    hipFree(d_fXsquared_two);
    hipFree(d_fYsquared_two);
    hipFree(d_snd_two);
    hipFree(d_error_two);
    hipSetDevice(0);

    free(snd);
    free(rec);
    free(halo);
    return 0;
}
